#include "hip/hip_runtime.h"
__global__
void rgba_to_greyscale(const uchar4* const rgbaImage, unsigned char* const greyImage, int numRows, int numCols)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
 	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

  	if (x < rows && y < cols) 
    	{ 
        	int idx = c + cols * r;
        	uchar4 pixel    = rgba[idx]; 
        	float intensity = 0.2126f * pixel.x + 0.7152f * pixel.y + 0.0722f * pixel.z; 
        	gray[idx]        = (unsigned char)intensity; 
    	} 

}

int main(int argc, char ** argv)
{
	if (argc > 2) 
    	{ 
        	imagePath = string(argv[1]); 
        	outputPath = string(argv[2]); 
    	} 

}
void your_rgba_to_greyscale(const uchar4 * const h_rgbaImage, uchar4 * const d_rgbaImage, unsigned char* const d_greyImage, size_t numRows, size_t numCols)
{
 	const dim3 blockSize(numCols/32, numCols/32 , 1);
  	const dim3 gridSize(numRows/12, numRows/12 , 1);
  	rgba_to_greyscale<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows, numCols);

  	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}