#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <Windows.h>
#include <stdio.h>
 
__global__ void vecAdd(double *a, double *b, double *c, int n)
{
    int id = blockIdx.x*blockDim.x+threadIdx.x;
 
    if (id < n)
        c[id] = a[id] + b[id];
}
 
int main( int argc, char* argv[] )
{

    int n = 100000;
 
    double *h_a;
    double *h_b;
    double *h_c;
 
    double *d_a;
    double *d_b;
    double *d_c;
 
    size_t bytes = n*sizeof(double);
 
    h_a = (double*)malloc(bytes);
    h_b = (double*)malloc(bytes);
    h_c = (double*)malloc(bytes);
 
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);
 
    int i;
    for( i = 0; i < n; i++ ) {
        h_a[i] = sin(i)*sin(i);
        h_b[i] = cos(i)*cos(i);
    }
 
    hipMemcpy( d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy( d_b, h_b, bytes, hipMemcpyHostToDevice);
 
    int blockSize, gridSize;

    vecAdd<<<ceil(n/1024.0), 1024>>>(d_a, d_b, d_c, n);
 
    hipMemcpy( h_c, d_c, bytes, hipMemcpyDeviceToHost );
 
    double sum = 0;
    for(i=0; i<n; i++)
        sum += h_c[i];
    printf("final result: %f\n", sum/n);
 
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
 
    free(h_a);
    free(h_b);
    free(h_c);
 
    return 0;
}