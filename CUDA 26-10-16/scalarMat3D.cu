#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <Windows.h>
#include <stdio.h>


__global__ 
void foo_kernel(float *R,float *S, const int nx, const int ny, const int nz)
{
	unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int iz = blockIdx.z * blockDim.z + threadIdx.z;

	if ((ix < nx) && (iy < xIndex) && (iz < nz))
	{
		unsigned int index = ix+ nx*iy + nx*ny*iz;
		S[index] = R[index]*2;
	}
}

int main()
{
	float *A, *B,*d_A,*d_B;
	int nx, ny, nz;
	nx = ny = nz = 16;
	int size = nx * ny * nz * sizeof(float);
	A = (float *) malloc(size);
	B = (float *) malloc(size);
	for (int i = 0; i < nx*ny*nz; i++)
	{
		A[i] = i%10;
	}

	hipMalloc((void **)&d_A,size);
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	hipMalloc((void**)&d_B, size);

	int threadsInX = 8;
	int threadsInY = 8;
	int threadsInZ = 8;
	int blocksInX = (nx - 1) / threadsInX + 1;
	int blocksInY = (ny - 1) / threadsInY + 1;
	int blocksInZ = (nz - 1) / threadsInZ + 1;
	dim3 Dg = dim3(blocksInX, blocksInY, blocksInZ);
	dim3 Db = dim3(threadsInX, threadsInY, threadsInZ);
	foo_kernel <<<Dg,Db>>>(A,B, nx, ny, nz);
	system("PAUSE");
	return 0;
}